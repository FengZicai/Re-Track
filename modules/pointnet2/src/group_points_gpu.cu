#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"

// input: grad_out (b,m,nsample,c), idx (b,m,nsample), 
// output: grad_points (b,n,c)
__global__ void group_points_grad_kernel_fast(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m*nsample*batch_index;
    grad_out += m*nsample*c*batch_index;
    grad_points += n*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                 atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
            }
        }
    }
}

void group_points_grad_kernel_launcher_fast(int b, int n, int c, int m, int nsample, 
    const float *grad_out, const int *idx, float *grad_points, hipStream_t stream) {
    // grad_out: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      grad_points: (B, C, N)
    hipError_t err;
    // dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    // dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_fast<<<b,256, 0, stream>>>(b,n,c,m,nsample,grad_out,idx,grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


// input: points (b,n,c), idx (b,m,nsample)
// output: out (b,m,nsample,c)
__global__ void group_points_kernel_fast(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
    }
}


void group_points_kernel_launcher_fast(int b, int n, int c, int m, int nsample, 
    const float *points, const int *idx, float *out, hipStream_t stream) {
    // points: (B, C, N)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints, nsample)
    hipError_t err;
    // dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    // dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_fast<<<b,256, 0, stream>>>(b,n,c,m,nsample,points,idx,out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
